#include "hip/hip_runtime.h"
#include "socialForce.cuh"
#include "socialForceHeader.cuh"
#include "gsimcore.cuh"
#include <fstream>
#include <iostream>
#include <string>
#include <sstream>
#include <iterator>
#include <iomanip>
#ifdef _WIN32
#include <Windows.h>
#include "gsimvisual.cuh"
#else
#include <sys/time.h>
#endif
#include "hip/hip_runtime.h"

void initOnDevice(float *x_pos, float *y_pos){
	float *x_pos_h, *y_pos_h;
	x_pos_h = (float*)malloc(AGENT_NO*sizeof(float));
	y_pos_h = (float*)malloc(AGENT_NO*sizeof(float));

	std::ifstream fin(dataFileName);
	std::string rec;

	char *cstr, *p;
	int i = 0;
	cstr = (char *)malloc(20 * sizeof(char));
	while (!fin.eof() && i<AGENT_NO) {
		std::getline(fin, rec);
		std::strcpy(cstr, rec.c_str());
		if(strcmp(cstr,"")==0)
			break;
		p=strtok(cstr, " ");
		x_pos_h[i] = atof(p);
		p=strtok(NULL, " ");
		y_pos_h[i] = atof(p);
		i++;
	}
	size_t floatDataSize = AGENT_NO*sizeof(float);
	hipMemcpy(x_pos, x_pos_h, floatDataSize, hipMemcpyHostToDevice);
	hipMemcpy(y_pos, y_pos_h, floatDataSize, hipMemcpyHostToDevice);
	getLastCudaError("initOnDevice");
}

__global__ void addAgentsOnDevice(SocialForceModel *sfModel){
	const int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < AGENT_NO_D){ // user init step
		//Add agent here
		SocialForceAgent *ag = new SocialForceAgent(idx, sfModel);
		sfModel->agentPool->ptrArray[idx] = ag;
		sfModel->agentPool->delMark[idx] = false;
	}
	if (idx == 0) {
		//set assignment
	}
}

void readConfig(char *config_file){
	std::ifstream fin;
	fin.open(config_file);
	std::string rec;
	char *cstr, *p;
	cstr = (char *)malloc(100 * sizeof(char));

	while (!fin.eof()) {
		std::getline(fin, rec);
		std::strcpy(cstr, rec.c_str());
		if(strcmp(cstr,"")==0)
			break;
		p=strtok(cstr, "=");
		if(strcmp(p, "AGENT_NO")==0){
			p=strtok(NULL, "=");
			AGENT_NO = atoi(p);
			hipMemcpyToSymbol(HIP_SYMBOL(AGENT_NO_D), &AGENT_NO, sizeof(int), 0, hipMemcpyHostToDevice);
			hipMemcpyToSymbol(HIP_SYMBOL(GLOBAL_ID), &AGENT_NO, sizeof(int), 0, hipMemcpyHostToDevice);
			getLastCudaError("readConfig");
		}
		if(strcmp(p, "MAX_AGENT_NO")==0){
			p=strtok(NULL, "=");
			MAX_AGENT_NO = atoi(p);
			hipMemcpyToSymbol(HIP_SYMBOL(MAX_AGENT_NO_D), &MAX_AGENT_NO, sizeof(int), 0, hipMemcpyHostToDevice);
			getLastCudaError("readConfig");
		}
		if(strcmp(p, "BOARDER_L")==0){
			p=strtok(NULL, "=");
			BOARDER_L_H = atoi(p);
			hipMemcpyToSymbol(HIP_SYMBOL(BOARDER_L_D), &BOARDER_L_H, sizeof(int));
			getLastCudaError("readConfig");
		}
		if(strcmp(p, "BOARDER_R")==0){
			p=strtok(NULL, "=");
			BOARDER_R_H = atoi(p);
			hipMemcpyToSymbol(HIP_SYMBOL(BOARDER_R_D), &BOARDER_R_H, sizeof(int));
			getLastCudaError("readConfig");
		}
		if(strcmp(p, "BOARDER_U")==0){
			p=strtok(NULL, "=");
			BOARDER_U_H = atoi(p);
			hipMemcpyToSymbol(HIP_SYMBOL(BOARDER_U_D), &BOARDER_U_H, sizeof(int));
			getLastCudaError("readConfig");
		}
		if(strcmp(p, "BOARDER_D")==0){
			p=strtok(NULL, "=");
			BOARDER_D_H = atoi(p);
			hipMemcpyToSymbol(HIP_SYMBOL(BOARDER_D_D), &BOARDER_D_H, sizeof(int));
			getLastCudaError("readConfig");
		}
		if(strcmp(p, "RANGE")==0){
			p=strtok(NULL, "=");
			RANGE_H = atof(p);
			hipMemcpyToSymbol(HIP_SYMBOL(RANGE), &RANGE_H, sizeof(float));
			getLastCudaError("readConfig");
		}
		if(strcmp(p, "DISCRETI")==0){
			p=strtok(NULL, "=");
			DISCRETI = atoi(p);
		}
		if(strcmp(p, "STEPS")==0){
			p=strtok(NULL, "=");
			STEPS = atoi(p);
		}
		if(strcmp(p, "VERBOSE")==0){
			p=strtok(NULL, "=");
			VERBOSE = atoi(p);
		}
		if(strcmp(p, "SELECTION")==0){
			p=strtok(NULL, "=");
			SELECTION = atoi(p);
		}
		if(strcmp(p, "VISUALIZE")==0){
			p=strtok(NULL, "=");
			VISUALIZE = atoi(p);
		}
		if(strcmp(p, "FILE_GEN")==0){
			p=strtok(NULL, "=");
			FILE_GEN = atoi(p);
		}
		if(strcmp(p, "BLOCK_SIZE")==0){
			p=strtok(NULL, "=");
			BLOCK_SIZE = atoi(p);
		}
		if(strcmp(p, "HEAP_SIZE")==0){
			p=strtok(NULL, "=");
			HEAP_SIZE = atoi(p);
		}
		if(strcmp(p, "STACK_SIZE")==0){
			p=strtok(NULL, "=");
			STACK_SIZE = atoi(p);
		}
		if(strcmp(p, "DATA_FILENAME")==0){
			dataFileName = new char[20];
			p=strtok(NULL, "=");
			strcpy(dataFileName, p);
		}
	}
	free(cstr);
	fin.close();

	int CNO_PER_DIM_H = (int)pow((float)2, DISCRETI);
	hipMemcpyToSymbol(HIP_SYMBOL(CNO_PER_DIM), &CNO_PER_DIM_H, sizeof(int));
	getLastCudaError("readConfig");
	
	CELL_NO = CNO_PER_DIM_H * CNO_PER_DIM_H;
	hipMemcpyToSymbol(HIP_SYMBOL(CELL_NO_D), &CELL_NO, sizeof(int));
	getLastCudaError("readConfig");
	
	float CLEN_X_H = (float)(BOARDER_R_H-BOARDER_L_H)/CNO_PER_DIM_H;
	float CLEN_Y_H = (float)(BOARDER_D_H-BOARDER_U_H)/CNO_PER_DIM_H;
	hipMemcpyToSymbol(HIP_SYMBOL(CLEN_X), &CLEN_X_H, sizeof(int));
	getLastCudaError("readConfig");
	hipMemcpyToSymbol(HIP_SYMBOL(CLEN_Y), &CLEN_Y_H, sizeof(int));
	getLastCudaError("readConfig");

	//GRID_SIZE = AGENT_NO%BLOCK_SIZE==0 ? AGENT_NO/BLOCK_SIZE : AGENT_NO/BLOCK_SIZE + 1;
}

void oneStep(SocialForceModel *model, SocialForceModel *model_h){
	AGENT_NO = model_h->agentPoolHost->numElem;
	SocialForceAgent **poolAgentList = model_h->agentPoolHost->ptrArray;
	GAgent **schAgentList = model_h->schedulerHost->allAgents;
	hipMemcpy(schAgentList, poolAgentList, AGENT_NO * sizeof(GAgent*), hipMemcpyDeviceToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(AGENT_NO_D), &AGENT_NO, sizeof(int), 0, hipMemcpyHostToDevice);
	if (model_h->worldHost != NULL) {
		GAgent **worldAgentList = model_h->worldHost->allAgents;
		hipMemcpy(worldAgentList, poolAgentList, AGENT_NO * sizeof(GAgent*), hipMemcpyDeviceToDevice);
	}

	GSimVisual::getInstance().animate();

	int gSize = GRID_SIZE(AGENT_NO);
	size_t sizeOfSmem = BLOCK_SIZE * (
		4*sizeof(int)
		+ sizeof(dataUnion)
		);

	getLastCudaError("before loop");
	util::genNeighbor(model_h->world, model_h->worldHost);
	getLastCudaError("end genNeighbor");
	step<<<gSize, BLOCK_SIZE, sizeOfSmem>>>(model);
	getLastCudaError("end step");	

	int scrGSize = GRID_SIZE(MAX_AGENT_NO);
	poolUtil::cleanup(model_h->agentPoolHost, model_h->agentPool);
}

void mainWork(char *config_file){
	hipDeviceSetCacheConfig(hipFuncCachePreferShared);
	getLastCudaError("setting cache preference");
	readConfig(config_file);

	size_t pVal;
	hipDeviceGetLimit(&pVal, hipLimitMallocHeapSize);
	printf("hipLimitMallocHeapSize: %d\n", pVal);
	hipDeviceSetLimit(hipLimitMallocHeapSize, HEAP_SIZE);
	hipDeviceGetLimit(&pVal, hipLimitMallocHeapSize);
	printf("hipLimitMallocHeapSize: %d\n", pVal);

	SocialForceModel *model = NULL;
	SocialForceModel *model_h = new SocialForceModel(BOARDER_D_H, BOARDER_D_H);
	util::copyHostToDevice(model_h, (void**)&model, sizeof(SocialForceModel));

	int gSize = GRID_SIZE(AGENT_NO);
	addAgentsOnDevice<<<gSize, BLOCK_SIZE>>>(model);
	getLastCudaError("before going into the big loop");
	printf("steps: %d\n", STEPS);
#ifdef _WIN32
	GSimVisual::getInstance().setWorld(model_h->world);
	for (int i=0; i<STEPS; i++){
		//if ((i%(STEPS/10))==0) 
		printf("STEP:%d ", i);
		oneStep(model, model_h);
	}
	printf("finally total agent is %d\n", AGENT_NO);
	GSimVisual::getInstance().stop();
#else
	for (int i=0; i<STEPS; i++){
	 	if ((i%(STEPS/10))==0) printf("STEP:%d ", i);
		oneStep(model, model_h);
		writeRandDebug(i, devRandDebug);
	}
#endif
	getLastCudaError("finished");
}

int main(int argc, char *argv[]){
#ifndef _WIN32
	struct timeval start, end;
	gettimeofday(&start, NULL);
	mainWork(argv[1]);
	gettimeofday(&end, NULL);
	printf("%ld\n", ((end.tv_sec * 1000000 + end.tv_usec)
		  - (start.tv_sec * 1000000 + start.tv_usec)));
#else
	int start = GetTickCount();
	mainWork(argv[1]);
	int end = GetTickCount();
	int diff = end-start;
	std::cout<<"Took "<<diff<<" ms"<<std::endl;
	system("PAUSE");
#endif
}
